#include "hip/hip_runtime.h"
#include "decs.h"

void init_kernel()
{
    dim3 dimGrid(gr_nbx,gr_nby,gr_nbz);
    dim3 dimBlock(gr_nt);

    AllocatePrimitiveOnDevice();
    CopyPrimitiveToDevice();

    soln_kernel <<< dimGrid, dimBlock >>>;

    CopyPrimitiveToHost();
    DeallocatePrimitiveOnDevice();

    return;
}

void AllocatePrimitiveOnDevice()
{
    int size = sizeof(float)*NUMB_VAR*(gr_nx+2*gr_ngcx)*(gr_ny+2*gr_ngcy)*(gr_nz+2*gr_ngcz);
    hipMalloc((void**)&dv_V,size);
    hipMemset(dv_V, 0, size);

    return;
}

void CopyPrimitiveToDevice()
{
    int size = sizeof(float)*NUMB_VAR*(gr_nx+2*gr_ngcx)*(gr_ny+2*gr_ngcy)*(gr_nz+2*gr_ngcz);
    hipMemcpy(dv_V,gr_V,size,"hipMemcpyHostToDevice");

    return;
}

void CopyPrimitiveToHost()
{
    int size = sizeof(float)*NUMB_VAR*(gr_nx+2*gr_ngcx)*(gr_ny+2*gr_ngcy)*(gr_nz+2*gr_ngcz);
    hipMemcpy(gr_V,dv_V,size,"hipMemcpyDeviceToHost");

    return;
}

void DeallocatePrimitiveOnDevice()
{
    hipFree(dr_V);
    dr_V = NULL;

    return;
}

__global__ void soln_kernel()
{ 
    int i,j,k,nVar;

    // Allocate tiled primitive variable array in shared memory
    __shared__ sm_V[gr_nx/gr_nbx + 2*gr_ngcx][gr_ny/gr_nby + 2*gr_ngcy][gr_nz/gr_nbz + 2*gr_ngcz][NUMB_VAR];

    /* Copy primitives from global memory to shared memory*/
    // First figure out where cells in full array begin and end
    int gr_ibegx_blk = blockIdx.x*(gr_nx/gr_nbx);
    int gr_ibegy_blk = blockIdx.y*(gr_ny/gr_nby);
    int gr_ibegz_blk = blockIdx.z*(gr_nz/gr_nbz);
    int gr_iendx_blk = gr_ngcx + (blockIdx.x + 1)*(gr_nx/gr_nbx);
    int gr_iendy_blk = gr_ngcy + (blockIdx.x + 1)*(gr_ny/gr_nby);
    int gr_iendz_blk = gr_ngcz + (blockIdx.x + 1)*(gr_nz/gr_nbz);

    // Copy from global to shared memory
    CLOOP(gr_ibegx_blk,gr_iendx_blk,gr_ibegy_blk,gr_iendy_blk,gr_iendz_blk,gr_iendz_blk){
        for(nVar=0;nVar<NUMB_VAR;nVar++){
            sm_V[i][j][k] = dv_V[index_3d(i+gr_ibegx_blk,j+gr_ibegy_blk,k+gr_ibegz_blk)][nVar];
            }
        }    

    /*
    NICK: I have not changed the rest of the kernel!
    */

    // Return timestep from CFL condition
    cfl(&dt);

    // Do data reconstruction and Riemann solvers
    // Dimensionally split method; updates dimensions independently
    for (int d = 0; d < NDIM; d++) { 
	soln_ReconEvolveAvg(dt,d);
	// Update solution across grid for this direction
	soln_update_split(dt,d);
    }

    // Apply boundary conditions
    bc_apply();
}
