#include "hip/hip_runtime.h"
#include "decs.h"

void init_kernel(float *dr_V, float *dr_VL, float *dr_VR, float *dr_flux)
{
    dim3 dimGrid(gr_nbx,gr_nby,gr_nbz);
    dim3 dimBlock(gr_nt,1,1);

    // DANAT: already done before
    // AllocatePrimitiveOnDevice();
    // CopyPrimitiveToDevice();

    cfl_kernel

    soln_kernel <<< dimGrid, dimBlock >>> (dr_V, dr_VL, dr_VR, dr_flux);

    bc_apply_kernel

    // CopyPrimitiveToHost();
    // DeallocatePrimitiveOnDevice();

    return;
}

void AllocatePrimitiveOnDevice()
{
    int size = sizeof(float)*NUMB_VAR*(gr_nx+2*gr_ngcx)*(gr_ny+2*gr_ngcy)*(gr_nz+2*gr_ngcz);
    hipMalloc((void**)&dv_V,size);
    hipMemset(dv_V, 0, size);

    return;
}

void CopyPrimitiveToDevice()
{
    int size = sizeof(float)*NUMB_VAR*(gr_nx+2*gr_ngcx)*(gr_ny+2*gr_ngcy)*(gr_nz+2*gr_ngcz);
    hipMemcpy(dv_V,gr_V,size,"hipMemcpyHostToDevice");

    return;
}

void CopyPrimitiveToHost()
{
    int size = sizeof(float)*NUMB_VAR*(gr_nx+2*gr_ngcx)*(gr_ny+2*gr_ngcy)*(gr_nz+2*gr_ngcz);
    hipMemcpy(gr_V,dv_V,size,"hipMemcpyDeviceToHost");

    return;
}

void DeallocatePrimitiveOnDevice()
{
    hipFree(dr_V);
    dr_V = NULL;

    return;
}

__global__ void soln_kernel(float *dr_V, float *dr_VL, float *dr_VR, float *dr_flux)
{
    int i,j,k,nVar;
    uint n = blockIdx.x * (blockDim.y * blockDim.z) + blockIdx.y * blockDim.z + blockIdx.z;

    // Allocate tiled primitive variable array in shared memory
    // DANAT: GPU version 1: not using shared memory; load everything from global
    // __shared__ sm_V[gr_nx/gr_nbx + 2*gr_ngcx][gr_ny/gr_nby + 2*gr_ngcy][gr_nz/gr_nbz + 2*gr_ngcz][NUMB_VAR];

    /* Copy primitives from global memory to shared memory*/
    // First figure out where cells in full array begin and end
    int gr_ibegx_blk = blockIdx.x * (gr_nx/gr_nbx);
    int gr_ibegy_blk = blockIdx.y * (gr_ny/gr_nby);
    int gr_ibegz_blk = blockIdx.z * (gr_nz/gr_nbz);
    int gr_iendx_blk = gr_ngcx + (blockIdx.x + 1) * (gr_nx/gr_nbx) - 1;
    int gr_iendy_blk = gr_ngcy + (blockIdx.x + 1) * (gr_ny/gr_nby) - 1;
    int gr_iendz_blk = gr_ngcz + (blockIdx.x + 1) * (gr_nz/gr_nbz) - 1;

    // Copy from global to shared memory
    // DANAT: don't forget about filling the guard cells too!
    // CLOOP(gr_ibegx_blk,gr_iendx_blk,gr_ibegy_blk,gr_iendy_blk,gr_iendz_blk,gr_iendz_blk){
    //     for(nVar=0;nVar<NUMB_VAR;nVar++){
    //             sm_V[i][j][k] = dv_V[index_3d(i+gr_ibegx_blk,j+gr_ibegy_blk,k+gr_ibegz_blk)][nVar];
    //         }
    // }

    // Return timestep from CFL condition
    // DANAT: for now I'll assume that dt is supplied externally; will fix it later
    // cfl(&dt);

    // Do data reconstruction and Riemann solvers
    // Dimensionally split method; updates dimensions independently
    for (int d = 0; d < NDIM; d++) {
    	soln_ReconEvolveAvg(dt, d, dr_V[n], dr_VL[n], dr_VR[n], dr_flux[n]);
    	// Update solution across grid for this direction
    	soln_update_split(dt, d, dr_V[n], dr_flux[n]);
    }

    // Apply boundary conditions
    // DANAT: Can we do it on CPU?
    // bc_apply(dr_V[n]);
}
